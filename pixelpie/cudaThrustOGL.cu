#include "hip/hip_runtime.h"
#include "cudaThrustOGL.hpp"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/unique.h>

#include <cassert>

typedef unsigned int uint;
typedef GLubyte mask_t;

// texture binding point for cuda access (for depth map)
texture<uchar1, hipTextureType2D, hipReadModeElementType> cudaTex;
__constant__ GLuint texwidth;

cudaThrustOGL::cudaThrustOGL() {
  err_ = hipDeviceReset();
  err_ = cudaGLSetGLDevice(0);
  err_ = hipSetDevice(0);
  assert(err_ == hipSuccess);
}

void cudaThrustOGL::cudaInit(const GLuint &texID, const GLuint &bufID,
                             const GLuint &emptybufID,
                             const GLuint &resultsBufID, const size_t &w,
                             const size_t &h) {
  width_ = w;
  height_ = h;

  // cuda register GL resources
  err_ = hipGraphicsGLRegisterImage(&cuda_res_[0], texID, GL_TEXTURE_2D,
                                     cudaGraphicsMapFlagsReadOnly);
  err_ = hipGraphicsGLRegisterBuffer(&cuda_res_[1], bufID,
                                      cudaGraphicsMapFlagsNone);
  err_ = hipGraphicsGLRegisterBuffer(&cuda_res_[2], emptybufID,
                                      cudaGraphicsMapFlagsNone);
  err_ = hipGraphicsGLRegisterBuffer(&cuda_res_[3], resultsBufID,
                                      cudaGraphicsMapFlagsNone);

  // upload the texture width to device
  GLuint uintw = (GLuint)width_;
  err_ = hipMemcpyToSymbol(HIP_SYMBOL(texwidth), &uintw, sizeof(uintw));
  if (err_ != hipSuccess) {
    std::cerr << "hipMemcpyToSymbol error" << err_ << " "
              << hipGetErrorName(err_) << std::endl;
  }

  seed_ = (unsigned int)time(NULL); // 12345

  reset();
  assert(err_ == hipSuccess);
}

void cudaThrustOGL::reset() {
  // init number of remaining darts to the size of the texture
  rem_darts_ = width_ * height_;
  iter_ = 0;
  rngoffset_ = 0; // keep using the random numbers
}

// Operator structs for counting empty pixels
struct isEmpty {
  __device__ bool operator()(const GLuint &i) {
    GLuint x = i % texwidth;
    GLuint y = i / texwidth;
    return (tex2D(cudaTex, x, y).x == 0);
  }
};

struct notEmpty {
  __device__ bool operator()(const GLuint &i) {
    GLuint x = i % texwidth;
    GLuint y = i / texwidth;
    return (tex2D(cudaTex, x, y).x != 0);
  }
};

size_t cudaThrustOGL::thrustCountEmptyPixels() {
  err_ = hipGraphicsMapResources(3, &cuda_res_[0]);

  // get the texture array
  hipArray *cuda_array;
  err_ = hipGraphicsSubResourceGetMappedArray(&cuda_array, cuda_res_[0], 0, 0);
  // bind the texture to cuda
  err_ = hipBindTextureToArray(cudaTex, cuda_array);

  // get the emptylist buffer
  GLuint *emptylistbuf;
  size_t bufsize;
  err_ = hipGraphicsResourceGetMappedPointer((void **)&emptylistbuf, &bufsize,
                                              cuda_res_[2]);

  // convert raw ptr to thrust ptr
  thrust::device_ptr<GLuint> dev_ptr =
      thrust::device_pointer_cast(emptylistbuf);

  // cout << "before thrust " << dev_ptr[0] << " " << dev_ptr[1] << endl;

  // declare the newend of the emptylist
  thrust::device_ptr<GLuint> newend;

  if (iter_ == 0) { // use counting itr to accelerate the first iteration
    newend = thrust::copy_if(thrust::make_counting_iterator<GLuint>(0),
                             thrust::make_counting_iterator<GLuint>(rem_darts_),
                             dev_ptr, isEmpty());
  } else {
    newend = thrust::remove_if(dev_ptr, dev_ptr + rem_darts_, notEmpty());
  }

  // cout << "after thrust " << dev_ptr[0] << " " << dev_ptr[1] << endl;

  err_ = hipUnbindTexture(cudaTex);
  err_ = hipGraphicsUnmapResources(3, &cuda_res_[0]);

  size_t newrem_darts = newend - dev_ptr;
  assert(newrem_darts < rem_darts_);

  rem_darts_ = newrem_darts;

  iter_++;
  assert(err_ == hipSuccess);
  return rem_darts_;
}

void cudaThrustOGL::cudaCleanup() {
  hipGraphicsUnmapResources(3, &cuda_res_[0]);
  for (size_t i = 0; i < 3; i++) {
    hipGraphicsUnregisterResource(cuda_res_[i]);
  }
  // hipFree(drandvec);
}

size_t cudaThrustOGL::freeGPUMem() {
  glFinish();
  err_ = hipDeviceSynchronize();
  size_t avail;
  size_t total;
  err_ = hipMemGetInfo(&avail, &total);
  // cout << "Device memory available: " << avail*1.0/1048576 << "MB" <<endl;
  assert(err_ == hipSuccess);
  return avail;
}

void cudaThrustOGL::remDuplicateSamples(size_t dartCount) {
  unsigned int *buf;
  size_t bufSize;
  err_ = hipGraphicsMapResources(1, &cuda_res_[3]);
  err_ = hipGraphicsResourceGetMappedPointer((void **)&buf, &bufSize,
                                              cuda_res_[3]);

  thrust::device_ptr<unsigned int> buf_first(buf);
  thrust::device_ptr<unsigned int> buf_last = buf_first + dartCount;
  thrust::device_ptr<unsigned int> buf_last_new =
      thrust::unique(buf_first, buf_last);

  hipGraphicsUnmapResources(1, &cuda_res_[3]);
}

// Random uniform distribution for transform
template <typename T> class random_uniform {
private:
  thrust::random::default_random_engine rng;
  // thrust::random::ranlux48 rng;
  thrust::uniform_real_distribution<float> dist;
  // Pointer to the empty list
  const T *empty_ptr_; //,*r_ptr_;

  // Offset for the rng
  const size_t offset_;

  // Number of elements in the empty list
  T umax_;

  // Size of the texture
  const size_t w_, h_;

  // scale normalized coord to ushort variables
  float wscale_, hscale_;

  // subpixel variables
  float sppixelw_, sprowarea_;

public:
  // provide constructor to initialize the distribution:
  random_uniform(const size_t &w, const size_t &h, const T &umax, const T *dp,
                 const size_t &o, const unsigned int &s)
      : dist(), empty_ptr_(dp), offset_(o), umax_(umax), w_(w), h_(h) {
    rng.seed(s);

    hscale_ = (USHRT_MAX * 1.0f / h_);
    wscale_ = (USHRT_MAX * 1.0f / w_);

    sppixelw_ = h_ * 1.0f / USHRT_MAX * w_ * 1.0f / USHRT_MAX;
    sprowarea_ = h_ * 1.0f / USHRT_MAX;
  }

  // OK, now the actual operator:
  __device__ T operator()(size_t index) {
    // skip past numbers used in previous threads
    rng.discard(index + offset_); // offset the used random numbers

    float fidx = dist(rng) * umax_;
    T coord = floor(fidx);
    float frac = fidx - floor(fidx);

    if (empty_ptr_ != NULL) {
      coord = empty_ptr_[coord]; // sample from the empty list
    }

    // pack the coordinates into ushorts
    GLushort x = (coord % w_) * wscale_;
    GLushort y = (coord / w_) * hscale_;

    // subpixel
    x += floor(fmod(frac, sprowarea_) / sppixelw_); // div subpix width
    y += floor(frac / sprowarea_);                  // div subpix row area

    // pack x y into uint
    T res = (y << 16 | x & 0xffff);
    return res;
  }
};

// Generate some vertices
void cudaThrustOGL::makeVertices(const size_t &ndarts) {
  err_ = hipGraphicsMapResources(3, &cuda_res_[0]);

  // get the dart buffer
  GLuint *dartbuf;
  size_t bufsize;
  err_ = hipGraphicsResourceGetMappedPointer((void **)&dartbuf, &bufsize,
                                              cuda_res_[1]);

  // get the emptylist buffer
  GLuint *emptylistbuf;
  err_ = hipGraphicsResourceGetMappedPointer((void **)&emptylistbuf, &bufsize,
                                              cuda_res_[2]);

  // convert raw ptr to thrust ptr
  thrust::device_ptr<GLuint> dart_ptr = thrust::device_pointer_cast(dartbuf);

  if (iter_ == 0) {
    emptylistbuf = NULL; // do not use the emptylist lookup in iter 0
  }

  // pick ndarts locations from the emptylist
  thrust::transform(thrust::make_counting_iterator<GLuint>(0),
                    thrust::make_counting_iterator<GLuint>(ndarts), dart_ptr,
                    random_uniform<GLuint>(width_, height_, rem_darts_ - 1,
                                           emptylistbuf, rngoffset_, seed_));
  rngoffset_ += ndarts;
  err_ = hipGraphicsUnmapResources(3, &cuda_res_[0]);
  assert(err_ == hipSuccess);
}
